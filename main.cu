#include "hip/hip_runtime.h"

#include <opencv2/opencv.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <iostream>

using namespace cv;
using namespace std;

// CUDA Kernel to convert image to grayscale
__global__ void rgb_to_gray_kernel(uchar3* rgb, uchar* gray, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        uchar3 pixel = rgb[idx];
        gray[idx] = 0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z;
    }
}

int main(int argc, char** argv) {
    if (argc != 2) {
        cout << "Usage: ./main <input_image_path>" << endl;
        return -1;
    }

    string input_image_path = argv[1];
    Mat img = imread(input_image_path, IMREAD_COLOR);

    if (img.empty()) {
        cout << "Error loading image." << endl;
        return -1;
    }

    int img_size = img.rows * img.cols;
    uchar3* d_rgb;
    uchar* d_gray;

    hipMalloc(&d_rgb, img_size * sizeof(uchar3));
    hipMalloc(&d_gray, img_size * sizeof(uchar));

    hipMemcpy(d_rgb, img.ptr<uchar3>(), img_size * sizeof(uchar3), hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((img.cols + block.x - 1) / block.x, (img.rows + block.y - 1) / block.y);
    rgb_to_gray_kernel<<<grid, block>>>(d_rgb, d_gray, img.cols, img.rows);

    Mat gray(img.rows, img.cols, CV_8UC1);
    hipMemcpy(gray.ptr<uchar>(), d_gray, img_size * sizeof(uchar), hipMemcpyDeviceToHost);

    hipFree(d_rgb);
    hipFree(d_gray);

    imwrite("grayscale_output.png", gray);
    cout << "Converted image saved as grayscale_output.png\n";

    return 0;
}
